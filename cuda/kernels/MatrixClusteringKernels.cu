#include "hip/hip_runtime.h"
#ifndef EUCLIDEAN_CLUSTERING_MATRIX_KERNEL_CU_
#define EUCLIDEAN_CLUSTERING_MATRIX_KERNEL_CU_

#define BLOCK_SIZE 128
#include "kernels/cuda_debug_utills.cu"
#include "include/MatrixClustering.cuh"

__device__ float point_distance(const CudaPoint a, const CudaPoint b) {
  return sqrt((a.x - b.x) * (a.x - b.x) +
      (a.y - b.y) * (a.y - b.y) +
      (a.z - b.z) * (a.z - b.z));
}

/**
 * performs initial clustering, called with 1D grid of 1D blocks.
 * @param cluster_cloud points and their lables
 * @param d_th euclidean clustering distance
 */
__global__ void initial_ec(CudaPointCloud cluster_cloud, float d_th) {
  unsigned tid = threadIdx.x;
  unsigned bid = blockIdx.x;
  unsigned b_top = min((unsigned)cluster_cloud.size - blockDim.x * blockIdx.x, blockDim.x);
  size_t gid = tid + bid * blockDim.x;  // get global thread id
  CudaPoint p{};
  if (gid < cluster_cloud.size){
    p = cluster_cloud.points[gid];
  }

  // allocate shared libraries
  __shared__ CudaPoint shared_points[BLOCK_SIZE];
  __shared__ unsigned labels[BLOCK_SIZE];
  __shared__ bool status[BLOCK_SIZE];

  shared_points[tid] = p;
  labels[tid] = tid;
  __syncthreads();
  for (int j = 0; j < b_top - 1; j++) {
    status[tid] = false;
    CudaPoint q = shared_points[j];
    unsigned cc = labels[tid];  // column cluster
    unsigned rc = labels[j];  // row cluster
    if (tid > j && point_distance(q, p) < d_th && rc != cc) {
      status[cc] = true;
    }
    __syncthreads();
    if (status[cc]) {
      labels[tid] = rc;
    }
    __syncthreads();
  }
  if(gid < cluster_cloud.size){
    cluster_cloud.labels[gid] = labels[tid] + bid * blockDim.x;
  }
}

namespace BuildMatrix {

__global__ void set_array_to_tid(DeviceArray<unsigned> array) {
  size_t global_id = threadIdx.x + blockIdx.x * blockDim.x;
  if (global_id >= array.size)
    return;
  array.data[global_id] = global_id;
}

__global__ void set_label_list(const CudaPointCloud cluster_cloud,
                               DeviceArray<unsigned> labels_pos) {
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= cluster_cloud.size)
    return;
  size_t cluster_id = cluster_cloud.labels[tid];
  if (cluster_id >= labels_pos.size)
    return;
  labels_pos.data[cluster_id] = 1;
}

__global__ void cluster_update(const CudaPointCloud cluster_cloud,
                               DeviceArray<unsigned> cluster_map) {

  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= cluster_cloud.size)
    return;
  cluster_cloud.labels[tid] = cluster_map.data[cluster_cloud.labels[tid]];
}
/**
 * Populates the adjacency matrix
 * @param cluster_cloud points and their labels
 * @param matrix adjacency matrix of clusters
 * @param d_th distance threshold between points
 */
__global__ void
populate_matrix(const CudaPointCloud cluster_cloud, DeviceMatrix<uint8_t> matrix, float d_th) {
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= cluster_cloud.size)  // check if thread is excess
    return;
  CudaPoint p = cluster_cloud.points[tid];
  for (int i = 0; i <= tid; i++) {
    CudaPoint q = cluster_cloud.points[i];
    if (point_distance(p, q) < d_th ) {
      size_t matrix_index = cluster_cloud.labels[tid] + cluster_cloud.labels[i] * matrix.step;
      matrix.matrix.data[matrix_index] = 1;
    }
  }
}

}  // namespace BuildMatrix

namespace MatrixMerge {

__device__ uint8_t getSubmatField(DeviceSubmatrixView<uint8_t> submat, MatrixPoint field) {
  unsigned offset_step = submat.origin_.x + submat.origin_.y * submat.parent_matrix_.step;
  unsigned field_step = field.x + field.y * submat.parent_matrix_.step;
  if (field_step + offset_step >= submat.parent_matrix_.size) {
    printf("Submat{%d,%d}: Error accessing (%d,%d)\n",
           submat.origin_.x,
           submat.origin_.y,
           field.x,
           field.y);
    return 255;
  }
  return submat.parent_matrix_.matrix.data[field_step + offset_step];
}


__device__
void
diagonal(DeviceSubmatrixView<uint8_t> submatrix,
         bool *was_merged,
         DeviceArray<unsigned> label_list) {
  unsigned tid = threadIdx.x;
  bool is_out_of_bound = tid > submatrix.step_-1;

  __shared__ unsigned labels[BLOCK_SIZE];
  __shared__ bool status[BLOCK_SIZE];
  bool thread_merged = false;
  labels[tid] = tid;
  __syncthreads();
  for (unsigned j = 0; j < submatrix.step_ - 1; j++) {
    status[tid] = false;
    __syncthreads();

    unsigned cc = labels[tid];  // current label of thread point
    unsigned rc = labels[j];  // current thread of compared point

    uint8_t label_status = (is_out_of_bound) ? 0 : getSubmatField(submatrix, {tid, j});
    if (tid > j && label_status == 1 && rc != cc) {
      status[cc] = true;
    }
    __syncthreads();

    if (status[cc]) {
      labels[tid] = rc;
      thread_merged = true;
      *was_merged = true;
    }
    __syncthreads();
  }
  if (thread_merged){
//    printf("changed %d to %d\n",label_list.data[tid + submatrix.submatrix_origin_.x], labels[tid]+ submatrix.submatrix_origin_.x);
    label_list.data[tid + submatrix.origin_.x] = labels[tid]+ submatrix.origin_.x;
  }
}

__device__ void offdiagonal(DeviceSubmatrixView<uint8_t> submatrix,
                            bool *was_merged,
                            DeviceArray<unsigned> label_list) {
  unsigned tid = threadIdx.x;
  unsigned submat_x = submatrix.origin_.x / BLOCK_SIZE;  // g1
  unsigned submat_y = submatrix.origin_.y / BLOCK_SIZE;  // g0

  unsigned cl = tid;
  bool merged_thread = false;
  __shared__ bool merged_block;
  __shared__ bool status[2 * BLOCK_SIZE];

  if (tid == 0) merged_block = false;
  __syncthreads();
  for (unsigned j = 0; j < submatrix.height_; j++) {
    status[tid] = false;
    status[tid + BLOCK_SIZE] = false;
    __syncthreads();


    uint8_t label_status = (tid >= submatrix.step_) ? 0 : getSubmatField(submatrix, {tid, j});
    if (label_status == 1) {
      status[cl] = true;

    }
    if (status[cl]) {
      cl = BLOCK_SIZE + j;
      merged_thread = true;
    }
    __syncthreads();
  }
  if (merged_thread) {
    label_list.data[tid + submatrix.origin_.x] = submatrix.origin_.y + cl - BLOCK_SIZE;
    merged_block = true;
  }
  __syncthreads();

  if(merged_block && tid == 0){
    *was_merged = true;
  }
}

__global__ void launchLayerMerge(DeviceSubmatrixView<uint8_t> *layer, bool *was_merged, DeviceArray<unsigned> label_list) {
  unsigned tid = threadIdx.x;
  unsigned bid = blockIdx.x;
//  printf("Launched submat {%d, %d}\n", layer[bid].submatrix_origin_.x,layer[bid].submatrix_origin_.y);
  if (layer[bid].origin_.x == layer[bid].origin_.y) {
    diagonal(layer[bid], was_merged, label_list);
  } else {
    offdiagonal(layer[bid], was_merged, label_list);
  }
}
}  // namespace MatrixMerge

namespace Update {
__global__ void mapClusters(CudaPointCloud cluster_cloud,
                            DeviceArray<unsigned> label_list,
                            DeviceArray<unsigned> label_map) {
  unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id >= cluster_cloud.size) {
    return;
  }
  cluster_cloud.labels[id] = label_list.data[label_map.data[cluster_cloud.labels[id]]];
}

__global__ void set_label_list(DeviceArray<unsigned> labels_list,
                               DeviceArray<unsigned> labels_pos) {
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= labels_list.size)
    return;
  size_t cluster_id = labels_list.data[tid];
  if (cluster_id >= labels_pos.size)
    return;
  labels_pos.data[cluster_id] = 1;
}

__global__ void build_matrix(DeviceMatrix<uint8_t> matrix,
                             DeviceMatrix<uint8_t> matrix_update,
                             DeviceArray<unsigned> label_list,
                             DeviceArray<unsigned> label_map) {
  unsigned column = threadIdx.x + blockIdx.x * blockDim.x;
  if (column >= matrix.step) {
    return;
  }

  unsigned column_updated = label_map.data[label_list.data[column]];
  for (unsigned row = 0; row < column; row++) {
    unsigned row_updated = label_map.data[label_list.data[row]];

    size_t matrix_index = row * matrix.step + column;
    if (matrix.matrix.data[matrix_index] == 1) {
      size_t matrix_update_index = row_updated * matrix_update.step + column_updated;

      matrix_update.matrix.data[matrix_update_index] = 1;
    }
  }
}
}

#endif //EUCLIDEAN_CLUSTERING_MATRIX_KERNEL_CU_