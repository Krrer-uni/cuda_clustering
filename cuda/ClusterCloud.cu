//
// Created by krrer on 04.11.23.
//
#include "include/ClusterCloud.cuh"
#include "include/CudaUtils.cuh"

void ClusterCloud::free() {
  if(points != nullptr){
    hipFree(points);
    hipDeviceSynchronize();
    cudaCheckError()
    points = nullptr;
  }
  if(labels != nullptr){
    hipFree(labels);
    hipDeviceSynchronize();
    cudaCheckError()
    labels = nullptr;
  }
}