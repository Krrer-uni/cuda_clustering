//
// Created by Wojciech Rymer on 04.11.23.
//
#include "include/CudaPointCloud.cuh"
#include "include/CudaUtils.cuh"

void CudaPointCloud::free() {
  if (points != nullptr) {
    hipFree(points);
    hipDeviceSynchronize();
    cudaCheckError()
    points = nullptr;
  }
  if (labels != nullptr) {
    hipFree(labels);
    hipDeviceSynchronize();
    cudaCheckError()
    labels = nullptr;
  }
}