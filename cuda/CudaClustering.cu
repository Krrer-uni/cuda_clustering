#include "hip/hip_runtime.h"
//
// Created by Wojciech Rymer on 04.11.23.
//
#include "include/MatrixClustering.cuh"
#include "kernels/MatrixClusteringKernels.cu"
#include "include/CudaUtils.cuh"
#include <pcl/point_types.h>
#include <cmath>
#include <thrust/scan.h>
template<class PointType>
MatrixClustering<PointType>::MatrixClustering() {
  cluster_cloud_ = std::make_shared<CudaPointCloud>();
  hipMalloc(&cluster_cloud_->points,12000);
  cluster_cloud_->free();
}

template<class PointType>
MatrixClustering<PointType>::~MatrixClustering() {
  cluster_cloud_->free();
  d_labels_map_.free();
  d_labels_list_.free();
}

template<class PointType>
void MatrixClustering<PointType>::setConfig(MatrixClusteringConfig config) {
  this->config_ = config;
}

template<class PointType>
void MatrixClustering<PointType>::setInputCloud(typename pcl::PointCloud<PointType>::Ptr input_cloud) {
  if (input_cloud == nullptr) {
    return;
  }
  cluster_cloud_->free();

  cluster_cloud_->size = input_cloud->size();
  size_t points_data_bytes = cluster_cloud_->size * sizeof(CudaPoint);
  size_t labels_data_bytes = cluster_cloud_->size * sizeof(unsigned);
  hipMallocManaged(&cluster_cloud_->points, points_data_bytes);
  hipDeviceSynchronize();
  cudaCheckError()
  hipMallocManaged(&cluster_cloud_->labels, labels_data_bytes);
  hipDeviceSynchronize();
  cudaCheckError()
  int j = 0;
  for(const auto &p : input_cloud->points){
    cluster_cloud_->points[j] = CudaPoint(p);
    j++;
  }
}
template<class PointType>
void MatrixClustering<PointType>::extract(std::vector<std::vector<int>> &indices_clusters) {

  // WARNING DATA ALLOCATED IN build_matrix()
  int grid_size = std::ceil(((float) cluster_cloud_->size) / BLOCK_SIZE);
  initial_ec<<<grid_size, BLOCK_SIZE>>>(*cluster_cloud_, config_.distance);
  hipDeviceSynchronize();
  cudaCheckError()

  build_matrix();
  hipDeviceSynchronize();
  cudaCheckError()

  bool main_loop = true;
  while (main_loop) {
    bool merge_found = false;

    bool is_final_layer = false;
    unsigned layer_count = 0;

    while (true) {
      auto layer = get_layer(layer_count, is_final_layer);
      layer_count++;
      if(layer.size() == 0)
        break;
      if (evaluate_layer(layer)) {
        update();
        break;
      }
      if (is_final_layer) {
        main_loop = false;
        break;
      }
    }
  }
  update();
//  std::cout << "number of labels: " << labels_list_.size << std::endl;
  indices_clusters.clear();
  indices_clusters.resize(d_labels_list_.size);
  for (size_t p = 0; p < cluster_cloud_->size; p++) {
    unsigned cl = cluster_cloud_->labels[p];
    indices_clusters[cl].push_back(p);
  }


  d_labels_list_.free();
  d_labels_map_.free();
  d_matrix_.free();
}

template<class PointType>
void MatrixClustering<PointType>::build_matrix() {
  d_labels_list_.free();
  d_labels_map_.free();

  d_labels_map_.size = cluster_cloud_->size + 1;  // n + 1 elements
  size_t labels_pos_data_size = d_labels_map_.size * sizeof(unsigned);
  hipMallocManaged(&d_labels_map_.data, labels_pos_data_size);
  hipDeviceSynchronize();
  hipMemset(d_labels_map_.data, 0u, labels_pos_data_size);
  hipDeviceSynchronize();
  cudaCheckError()

  int grid_size = std::ceil(((float) cluster_cloud_->size) / BLOCK_SIZE);
  BuildMatrix::set_label_list<<<grid_size, BLOCK_SIZE>>>(*cluster_cloud_, d_labels_map_);
  hipDeviceSynchronize();
  cudaCheckError()

  thrust::exclusive_scan(d_labels_map_.data, d_labels_map_.data + d_labels_map_.size, d_labels_map_.data);

  cudaCheckError()
  // R array allocation
  unsigned unique_clusters = d_labels_map_.data[d_labels_map_.size - 1];
  d_labels_list_.size = unique_clusters;
  hipMallocManaged(&d_labels_list_.data, d_labels_list_.size * sizeof(unsigned));
  BuildMatrix::set_array_to_tid<<<grid_size, BLOCK_SIZE>>>(d_labels_list_);
  hipDeviceSynchronize();
  cudaCheckError()

  BuildMatrix::cluster_update<<<grid_size, BLOCK_SIZE>>>(*cluster_cloud_, d_labels_map_);
  hipDeviceSynchronize();
  cudaCheckError()

  hipMemset(d_labels_map_.data, 0u, labels_pos_data_size);
  hipDeviceSynchronize();
  cudaCheckError()
  BuildMatrix::set_label_list<<<grid_size, BLOCK_SIZE>>>(*cluster_cloud_, d_labels_map_);
  hipDeviceSynchronize();
  cudaCheckError()
  thrust::exclusive_scan(d_labels_map_.data, d_labels_map_.data + d_labels_map_.size, d_labels_map_.data);

  d_matrix_.allocateMatrixZero(unique_clusters * unique_clusters);
  size_t matrix_grid_size = std::ceil(((float) d_matrix_.size) / BLOCK_SIZE);
  BuildMatrix::populate_matrix<<<matrix_grid_size, BLOCK_SIZE>>>(*cluster_cloud_,
      d_matrix_,
      config_.distance);
  hipDeviceSynchronize();
  cudaCheckError()
}

/**
 * Evaluate a layer of matrix
 * @tparam PointType Point type of input cloud
 * @param layer vector of submatricies belonging to one layer
 * @return true if a merged occurred, false otherwise
 */
template<class PointType>
bool MatrixClustering<PointType>::evaluate_layer(std::vector<DeviceSubmatrixView<uint8_t>>& layer) {
  bool* merge_found;  // variable in managed memory
  hipMallocManaged(&merge_found,sizeof merge_found);
  *merge_found = false;
  // allocate and copy submatrices to device memeory
  DeviceSubmatrixView<uint8_t>* d_layer;
  size_t layer_size = layer.size() * sizeof(DeviceSubmatrixView<uint8_t>) ;
  hipMallocManaged(&d_layer, layer_size);
  hipMemcpy(d_layer, layer.data(), layer_size, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  cudaCheckError()

  MatrixMerge::launchLayerMerge<<<layer.size(),BLOCK_SIZE>>>(d_layer,merge_found,d_labels_list_);
  hipDeviceSynchronize();
  cudaCheckError()

  hipFree(d_layer);
  hipDeviceSynchronize();
  cudaCheckError()
  return *merge_found;
}
/**
 *  Function to get submatrices of a layer
 * @tparam PointType Point type of input cloud
 * @param layer_number index of a layer
 * @param _is_last idicator if the returned layer is the last one
 * @return vector of submatricies belonging to one layer
 */
template<class PointType>
std::vector<DeviceSubmatrixView<uint8_t>> MatrixClustering<PointType>::get_layer(size_t layer_number, bool &_is_last) {
  std::vector<DeviceSubmatrixView<uint8_t>> layer{};
  for (unsigned x = layer_number * BLOCK_SIZE; x < d_labels_list_.size; x += BLOCK_SIZE) {
    unsigned y = x - layer_number * BLOCK_SIZE;
    unsigned blockWidth = std::min((size_t) BLOCK_SIZE, d_labels_list_.size - x);
    unsigned blockHeight = std::min((size_t) BLOCK_SIZE, d_labels_list_.size - y);
    layer.emplace_back(d_matrix_, MatrixPoint{x, y}, blockWidth, blockHeight);
  }
  if(layer.size() == 1)
    _is_last = true;
  return layer;
}


template<class PointType>
void MatrixClustering<PointType>::update() {
  int cloud_grid_size = std::ceil(((float) cluster_cloud_->size) / BLOCK_SIZE);
  Update::mapClusters<<<cloud_grid_size, BLOCK_SIZE>>>(*cluster_cloud_, d_labels_list_,d_labels_map_);
  hipDeviceSynchronize();
  cudaCheckError()
  size_t labels_pos_data_size = d_labels_map_.size * sizeof(unsigned);
  hipMemset(d_labels_map_.data, 0u, labels_pos_data_size);
  hipDeviceSynchronize();
  cudaCheckError()

  int label_grid_size = std::ceil(((float) d_labels_map_.size) / BLOCK_SIZE);
  Update::set_label_list<<<label_grid_size,BLOCK_SIZE>>>(d_labels_list_, d_labels_map_);
  hipDeviceSynchronize();
  cudaCheckError()
  thrust::exclusive_scan(d_labels_map_.data, d_labels_map_.data + d_labels_map_.size, d_labels_map_.data);

  unsigned unique_clusters = d_labels_map_.data[d_labels_map_.size - 1];
//  std::cout << "Found " << unique_clusters << " unique labels\n" ;
  DeviceArray<unsigned> labels_list_update{};
  labels_list_update.size = unique_clusters;
  hipMallocManaged(&labels_list_update.data, labels_list_update.size * sizeof(unsigned));
  hipDeviceSynchronize();
  cudaCheckError()

  BuildMatrix::set_array_to_tid<<<unique_clusters, BLOCK_SIZE>>>(labels_list_update);
  hipDeviceSynchronize();
  cudaCheckError()

  DeviceMatrix<uint8_t> matrix_update{};
  matrix_update.allocateMatrixZero(unique_clusters * unique_clusters);

  Update::build_matrix<<<label_grid_size, BLOCK_SIZE>>>(d_matrix_,matrix_update,d_labels_list_,d_labels_map_);
  hipDeviceSynchronize();
  cudaCheckError()

  d_labels_list_.free();
  d_labels_list_ = labels_list_update;

  d_matrix_.free();
  d_matrix_ = matrix_update;
}

/*
 * declarations of used templates
 */
template
class MatrixClustering<pcl::PointXYZ>;

template
class MatrixClustering<pcl::PointXYZI>;
