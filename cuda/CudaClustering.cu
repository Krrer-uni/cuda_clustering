#include "hip/hip_runtime.h"
#include "include/CudaClustering.cuh"
#include "kernels/matrix_clustering.cu"
#include <pcl/point_types.h>
#include "include/CudaUtils.cuh"
#include <cmath>

template<class PointType>
CudaClustering<PointType>::CudaClustering() {
  cluster_cloud_ = std::make_shared<ClusterCloud>();
}

template<class PointType>
CudaClustering<PointType>::~CudaClustering() {
  if (cluster_cloud_ != nullptr && cluster_cloud_->points != nullptr) {
    hipFree(cluster_cloud_->points);
  }
  if (cluster_cloud_ != nullptr && cluster_cloud_->labels != nullptr) {
    hipFree(cluster_cloud_->labels);
  }
}

template<class PointType>
void CudaClustering<PointType>::setParams(ClusterParams params) {
  this->params_ = params;
}

template<class PointType>
void CudaClustering<PointType>::setInputCloud(typename pcl::PointCloud<PointType>::Ptr input_cloud) {
  if (input_cloud == nullptr) {
    return;
  }

  if (cluster_cloud_ != nullptr && cluster_cloud_->points != nullptr) {
    hipFree(cluster_cloud_->points);
  }
  if (cluster_cloud_ != nullptr && cluster_cloud_->labels != nullptr) {
    hipFree(cluster_cloud_->labels);
  }

  cluster_cloud_->size = input_cloud->size();
  size_t points_data_bytes = cluster_cloud_->size * sizeof(CudaPoint);
  size_t labels_data_bytes = cluster_cloud_->size * sizeof(unsigned);
  hipMallocManaged(&cluster_cloud_->points, points_data_bytes);
  hipDeviceSynchronize();
  cudaCheckError()
  hipMallocManaged(&cluster_cloud_->labels, labels_data_bytes);
  hipDeviceSynchronize();
  cudaCheckError()

  for (size_t i = 0; i < input_cloud->size(); i++) {
    cluster_cloud_->points[i] = {input_cloud->points.data()[i].x,
                                 input_cloud->points.data()[i].y,
                                 input_cloud->points.data()[i].z};
  }
}

template<class PointType>
void CudaClustering<PointType>::extract(std::vector<unsigned int> &indices_clusters) {

  // WARNING DATA ALLOCATED IN build_matrix()
  labels_list_.data = nullptr;

  int grid_size = std::ceil(((float) cluster_cloud_->size) / BLOCK_SIZE);
  initial_ec<<<grid_size, BLOCK_SIZE>>>(*cluster_cloud_, params_.distance);

  build_matrix(*cluster_cloud_, labels_list_, d_matrix_, params_.distance);
//
//  for(int i = 0; i < d_matrix_.matrix_step; i++){
//    for(int j = 0; j < d_matrix_.matrix_step; j++){
//      std::cout << (unsigned)d_matrix_.matrix.data[i * d_matrix_.matrix_step + j] << " ";
//    }
//    std::cout << std::endl;
//  }


  bool main_loop = true;
  while (main_loop) {
    bool merge_found = false;

    bool is_final_layer = false;
    unsigned layer_count = 0;

    while (!merge_found) {
      auto layer = get_layer(layer_count, is_final_layer);
      layer_count++;
      if(layer.size() == 0)
        return;
      merge_found = merge_found || evaluate_layer(layer);
      if (merge_found) {
        update();
        continue;
      }
      if (is_final_layer) {
        main_loop = false;
        continue;
      }
    }
  }

  size_t labels_data_bytes = cluster_cloud_->size * sizeof(unsigned);
  hipMemcpy(indices_clusters.data(),
             cluster_cloud_->labels,
             labels_data_bytes,
             hipMemcpyDeviceToHost);

  if (labels_list_.data != nullptr) {
    hipFree(&labels_list_.data); // ALLOCATED IN build_matrix()
  }
  if (d_matrix_.matrix.data != nullptr) {
    hipFree(&d_matrix_.matrix); // ALLOCATED IN build_matrix()
  }
}

template<class PointType>
void CudaClustering<PointType>::build_matrix(ClusterCloud &cluster_cloud,
                                             DeviceArray<unsigned int> &labels_list,
                                             Matrix<uint8_t> &matrix,
                                             const float d_th) {
  if (labels_list.data != nullptr)
    hipFree(&labels_list.data);
  if (labels_map_.data != nullptr)
    hipFree(&labels_map_.data);
  matrix.free();

  labels_map_.size = cluster_cloud.size + 1;  // n + 1 elements
  size_t labels_pos_data_size = labels_map_.size * sizeof(unsigned);
  hipMallocManaged(&labels_map_.data, labels_pos_data_size);
  hipMemset(labels_map_.data, 0u, labels_pos_data_size);
  cudaCheckError()

  int grid_size = std::ceil(((float) cluster_cloud.size) / BLOCK_SIZE);
  BuildMatrix::set_label_list<<<grid_size, BLOCK_SIZE>>>(cluster_cloud, labels_map_);
  hipDeviceSynchronize();
  cudaCheckError()

  exclusive_scan(labels_map_);
  cudaCheckError()
  // R array allocation
  unsigned unique_clusters = labels_map_.data[labels_map_.size - 1];
  std::cout << "Found " << unique_clusters << " unique labels\n";
  labels_list.size = unique_clusters;
  hipMallocManaged(&labels_list.data, labels_list.size * sizeof(unsigned));
  BuildMatrix::set_array_to_tid<<<grid_size, BLOCK_SIZE>>>(labels_list);
  hipDeviceSynchronize();
  cudaCheckError()

  BuildMatrix::cluster_update<<<grid_size, BLOCK_SIZE>>>(cluster_cloud, labels_map_);
  hipDeviceSynchronize();
  cudaCheckError()

  matrix.allocateMatrixZero(unique_clusters * unique_clusters);
  size_t matrix_grid_size = std::ceil(((float) matrix.size) / BLOCK_SIZE);
  BuildMatrix::populate_matrix<<<matrix_grid_size, BLOCK_SIZE>>>(cluster_cloud,
      matrix,
      d_th);
  hipDeviceSynchronize();
  cudaCheckError()
}


template<class PointType>
void CudaClustering<PointType>::exclusive_scan(DeviceArray<unsigned> &array) {
  if(array.size == 0){
    return;
  }
  unsigned last_elem = array.data[0];

  array.data[0] = 0;
  for (size_t i = 1; i < array.size; i++) {
    unsigned tmp = array.data[i];
    array.data[i] = last_elem + array.data[i - 1];
    last_elem = tmp;
  }
}

template<class PointType>
bool CudaClustering<PointType>::evaluate_layer(std::vector<SubmatrixView<uint8_t>>& layer) {
  bool* merge_found;
  hipMallocManaged(&merge_found,sizeof merge_found);
  *merge_found = false;
  SubmatrixView<uint8_t>* d_layer;
  size_t layer_size = layer.size() * sizeof(SubmatrixView<uint8_t>) ;
  hipMallocManaged(&d_layer, layer_size);
  hipMemcpy(d_layer, layer.data(), layer_size, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  cudaCheckError()
  for(auto i = labels_list_.data; i < labels_list_.data + labels_list_.size; i++ ){
    std::cout << *i << " ";
  }
  std::cout << std::endl;
  MatrixMerge::launchLayerMerge<<<layer.size(),BLOCK_SIZE>>>(d_layer,merge_found,labels_list_);
  hipDeviceSynchronize();
  cudaCheckError()
  for(auto i = labels_list_.data; i < labels_list_.data + labels_list_.size; i++ ){
    std::cout << *i << " ";
  }
  std::cout << std::endl;
  hipFree(d_layer);
  return *merge_found;
}

template<class PointType>
std::vector<SubmatrixView<uint8_t>> CudaClustering<PointType>::get_layer(size_t layer_number, bool &_is_last) {
  std::vector<SubmatrixView<uint8_t>> layer{};
  for (unsigned x = layer_number * BLOCK_SIZE; x < labels_list_.size; x += BLOCK_SIZE) {
    unsigned y = x - layer_number * BLOCK_SIZE;
    unsigned blockWidth = std::min((size_t) BLOCK_SIZE, labels_list_.size - x);
    unsigned blockHeight = std::min((size_t) BLOCK_SIZE, labels_list_.size - y);
    layer.emplace_back(d_matrix_, MatrixPoint{x, y}, blockHeight, blockWidth);
  }
  if(layer.size() == 1)
    _is_last = true;
  return layer;
}


template<class PointType>
void CudaClustering<PointType>::update() {

  int cloud_grid_size = std::ceil(((float) cluster_cloud_->size) / BLOCK_SIZE);
  Update::mapClusters<<<cloud_grid_size, BLOCK_SIZE>>>(*cluster_cloud_, labels_list_,labels_map_);
  hipDeviceSynchronize();
  cudaCheckError()

  size_t labels_pos_data_size = labels_map_.size * sizeof(unsigned);
  hipMemset(labels_map_.data, 0u, labels_pos_data_size);
  hipDeviceSynchronize();
  cudaCheckError()

  int label_grid_size = std::ceil(((float) labels_map_.size) / BLOCK_SIZE);
  Update::set_label_list<<<label_grid_size,BLOCK_SIZE>>>(labels_list_, labels_map_);
  hipDeviceSynchronize();
  cudaCheckError()

  exclusive_scan(labels_map_);

  unsigned unique_clusters = labels_map_.data[labels_map_.size - 1];
  std::cout << "Found " << unique_clusters << " unique labels\n" ;
  DeviceArray<unsigned> labels_list_update{};
  labels_list_update.size = unique_clusters;
  hipMallocManaged(&labels_list_update.data, labels_list_update.size * sizeof(unsigned));
  hipDeviceSynchronize();
  cudaCheckError()

  BuildMatrix::set_array_to_tid<<<unique_clusters, BLOCK_SIZE>>>(labels_list_update);
  hipDeviceSynchronize();
  cudaCheckError()

  Matrix<uint8_t> matrix_update{};
  matrix_update.allocateMatrixZero(unique_clusters * unique_clusters);

  Update::build_matrix<<<label_grid_size, BLOCK_SIZE>>>(d_matrix_,matrix_update,labels_list_,labels_map_);
  hipDeviceSynchronize();
  cudaCheckError()

  hipFree(labels_list_.data);
  labels_list_ = labels_list_update;

  d_matrix_.free();
  d_matrix_ = matrix_update;
}

/*
 * declarations of used templates
 */
template
class CudaClustering<pcl::PointXYZ>;

template
class CudaClustering<pcl::PointXYZI>;
