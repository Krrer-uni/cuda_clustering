#include "include/DeviceArray.cuh"
#include "include/CudaUtils.cuh"

template<typename T>
void DeviceArray<T>::free() {
  if(data != nullptr){
    hipFree(data);
    hipDeviceSynchronize();
    cudaCheckError()
    data = nullptr;
  }
}

template class DeviceArray<unsigned>;