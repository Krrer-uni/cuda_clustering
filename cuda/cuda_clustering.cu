#include "hip/hip_runtime.h"
#include "cstdio"
#include "cuda_clustering.cuh"

#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

__global__ void cuda_hello() {
  auto n = threadIdx.x;
  printf("Hello from thread %d!!\n", n);
}

void cuda_wrapper(int n){
  cuda_hello<<<1,n>>>();
  hipDeviceSynchronize();
  cudaCheckError()
}
